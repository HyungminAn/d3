#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   https://www.lammps.org/, Sandia National Laboratories
   LAMMPS development team: developers@lammps.org

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing author: Hyungmin An (andynn@snu.ac.kr)
------------------------------------------------------------------------- */

#include "pair_d3.h"
#include <hip/hip_runtime.h>

using namespace LAMMPS_NS;

// linij and idx_linij -> limit=46340 atoms
inline __host__ __device__ void ij_at_linij(int linij, int &i, int &j) {
    i = (sqrtf(1 + 8.0f * linij) - 1) / 2; // static_cast or floor function or implicit
    j = linij - i * (i + 1) / 2;
}

// from MathExtra::lensq3
inline __host__ __device__ double lensq3(const double *v)
{
  return v[0] * v[0] + v[1] * v[1] + v[2] * v[2];
}


/* ----------------------------------------------------------------------
   Constructor (Required)
------------------------------------------------------------------------- */

PairD3::PairD3(LAMMPS* lmp) : Pair(lmp) {
    single_enable = 0;      // potential is not pair-wise additive.
    restartinfo = 0;        // Many-body potentials are usually not
                            // written to binary restart files.
    one_coeff = 1;          // Many-body potnetials typically read all
                            // parameters from a file, so only one
                            // pair_coeff statement is needed.
    manybody_flag = 1;
}

/* ----------------------------------------------------------------------
   Destructor (Required)
------------------------------------------------------------------------- */

PairD3::~PairD3() {
    if (allocated) {

        int n = atom->natoms;
        int np1 = atom->ntypes + 1;

        hipFree(r2r4);
        hipFree(rcov);
        hipFree(mxc);
        for (int i = 0; i < np1; i++) { hipFree(setflag[i]); }; hipFree(setflag);
        for (int i = 0; i < np1; i++) { hipFree(cutsq[i]); }; hipFree(cutsq);
        for (int i = 0; i < np1; i++) { hipFree(r0ab[i]); }; hipFree(r0ab);

        for (int i = 0; i < np1; i++) {
            for (int j = 0; j < np1; j++) {
                for (int k = 0; k < MAXC; k++) {
                    for (int l = 0; l < MAXC; l++) {
                        hipFree(c6ab[i][j][k][l]);
                    }
                    hipFree(c6ab[i][j][k]);
                }
                hipFree(c6ab[i][j]);
            }
            hipFree(c6ab[i]);
        }
        hipFree(c6ab);

        hipFree(lat_v_1);
        hipFree(lat_v_2);
        hipFree(lat_v_3);
        hipFree(cn);
        hipFree(dc6i);

        for (int i = 0; i < n; i++) { hipFree(x[i]); }; hipFree(x);
        for (int i = 0; i < n; i++) { hipFree(f[i]); }; hipFree(f);
        for (int i = 0; i < 3; i++) { hipFree(sigma[i]); }; hipFree(sigma);

        hipFree(dc6_iji_tot);
        hipFree(dc6_ijj_tot);
        hipFree(c6_ij_tot);
        hipFree(rep_vdw);
        hipFree(rep_cn);
    }
}

/* ----------------------------------------------------------------------
   Allocate all arrays (Required)
------------------------------------------------------------------------- */

void PairD3::allocate() {
    allocated = 1;

    /* atom->ntypes : # of elements; element index starts from 1 */
    int n = atom->natoms;
    int np1 = atom->ntypes + 1;
    n_save = n;

    hipMallocManaged(&setflag, np1 * sizeof(int*)); for (int i = 0; i < np1; i++) { hipMallocManaged(&setflag[i], np1 * sizeof(int)); }
    hipMallocManaged(&cutsq, np1 * sizeof(double*)); for (int i = 0; i < np1; i++) { hipMallocManaged(&cutsq[i], np1 * sizeof(double)); }
    hipMallocManaged(&r2r4, np1 * sizeof(double));
    hipMallocManaged(&rcov, np1 * sizeof(double));
    hipMallocManaged(&mxc, np1 * sizeof(int));
    hipMallocManaged(&r0ab, np1 * sizeof(double*)); for (int i = 0; i < np1; i++) { hipMallocManaged(&r0ab[i], np1 * sizeof(double)); }
    
    hipMallocManaged(&c6ab, np1 * sizeof(double****));
    for (int i = 0; i < np1; i++) {
        hipMallocManaged(&c6ab[i], np1 * sizeof(double***));
        for (int j = 0; j < np1; j++) {
            hipMallocManaged(&c6ab[i][j], MAXC * sizeof(double**));
            for (int k = 0; k < MAXC; k++) {
                hipMallocManaged(&c6ab[i][j][k], MAXC * sizeof(double*));
                for (int l = 0; l < MAXC; l++) {
                    hipMallocManaged(&c6ab[i][j][k][l], 3 * sizeof(double));
                }
            }
        }
    }

    hipMallocManaged(&lat_v_1, 3 * sizeof(double));
    hipMallocManaged(&lat_v_2, 3 * sizeof(double));
    hipMallocManaged(&lat_v_3, 3 * sizeof(double));
    hipMallocManaged(&rep_vdw, 3 * sizeof(int));
    hipMallocManaged(&rep_cn,  3 * sizeof(int));
    hipMallocManaged(&sigma,   3 * sizeof(double*)); for (int i = 0; i < 3; i++) { hipMallocManaged(&sigma[i], 3 * sizeof(double)); }

    hipMallocManaged(&cn, n * sizeof(double));
    hipMallocManaged(&x, n * sizeof(double*)); for (int i = 0; i < n; i++) { hipMallocManaged(&x[i], 3 * sizeof(double)); }
    hipMallocManaged(&dc6i, n * sizeof(double));
    hipMallocManaged(&f, n * sizeof(double*)); for (int i = 0; i < n; i++) { hipMallocManaged(&f[i], 3 * sizeof(double)); }

    // Initialization (by function)
    set_lattice_vectors();

    // Initialization
    for (int i = 1; i < np1; i++) {
        for (int j = 1; j < np1; j++) {
            setflag[i][j] = 0;
        }
    }

    for (int idx1 = 0; idx1 < np1;  idx1++) {
        for (int idx2 = 0; idx2 < np1;  idx2++) {
            for (int idx3 = 0; idx3 < MAXC; idx3++) {
                for (int idx4 = 0; idx4 < MAXC; idx4++) {
                    for (int idx5 = 0; idx5 < 3;    idx5++) {
                        c6ab[idx1][idx2][idx3][idx4][idx5] = -1;
                    }
                }
            }
        }
    }

    int n_ij_combination = n * (n + 1) / 2;
    hipMallocManaged(&dc6_iji_tot, n_ij_combination * sizeof(double));
    hipMallocManaged(&dc6_ijj_tot, n_ij_combination * sizeof(double));
    hipMallocManaged(&c6_ij_tot,   n_ij_combination * sizeof(double));

    //int vdw_range_x = 2 * rep_vdw[0] + 1;
    //int vdw_range_y = 2 * rep_vdw[1] + 1;
    //int vdw_range_z = 2 * rep_vdw[2] + 1;
    //tau_idx_vdw_total_size = vdw_range_x * vdw_range_y * vdw_range_z * 3;

    //int cn_range_x  = 2 * rep_cn[0] + 1;
    //int cn_range_y  = 2 * rep_cn[1] + 1;
    //int cn_range_z  = 2 * rep_cn[2] + 1;
    //tau_idx_cn_total_size = cn_range_x * cn_range_y * cn_range_z * 3;

    //hipMallocManaged(&tau_vdw, vdw_range_x * sizeof(double***));
    //for (int i = 0; i < vdw_range_x; i++) {
    //    hipMallocManaged(&tau_vdw[i], vdw_range_y * sizeof(double**));
    //    for (int j = 0; j < vdw_range_y; j++) {
    //        hipMallocManaged(&tau_vdw[i][j], vdw_range_z * sizeof(double*));
    //        for (int k = 0; k < vdw_range_z; k++) {
    //            hipMallocManaged(&tau_vdw[i][j][k], 3 * sizeof(double));
    //        }
    //    }
    //}
    //hipMallocManaged(&tau_idx_vdw, tau_idx_vdw_total_size * sizeof(int));

    //hipMallocManaged(&tau_cn, cn_range_x * sizeof(double***));
    //for (int i = 0; i < cn_range_x; i++) {
    //    hipMallocManaged(&tau_cn[i], cn_range_y * sizeof(double**));
    //    for (int j = 0; j < cn_range_y; j++) {
    //        hipMallocManaged(&tau_cn[i][j], cn_range_z * sizeof(double*));
    //        for (int k = 0; k < cn_range_z; k++) {
    //            hipMallocManaged(&tau_cn[i][j][k], 3 * sizeof(double));
    //        }
    //    }
    //}
    //hipMallocManaged(&tau_idx_cn, tau_idx_cn_total_size * sizeof(int));

}

/* ----------------------------------------------------------------------
   Settings: read from pair_style (Required)
             pair_style   d3 rthr cn_thr damping_type
------------------------------------------------------------------------- */

void PairD3::settings(int narg, char **arg) {
    if (narg != 3) {
        error->all(FLERR,
                "Pair_style d3 needs Three arguments:\n"
                "\t rthr : threshold for dispersion interaction\n"
                "\t cn_thr : threshold for coordination number calculation\n"
                "\t damping_type : type of damping function\n"
                );
    }
    rthr   = utils::numeric(FLERR, arg[0], false, lmp);
    cn_thr = utils::numeric(FLERR, arg[1], false, lmp);

    std::unordered_map<std::string, int> commandMap = {
        { "d3_damp_zero", 1}, { "d3_damp_bj", 2 },
        { "d3_damp_zerom", 3 }, { "d3_damp_bjm", 4 },
    };

    int commandCode = commandMap[arg[2]];
    switch (commandCode) {
    case 1: damping_type = 1; break;
    case 2: damping_type = 2; break;
    case 3: damping_type = 3; break;
    case 4: damping_type = 4; break;
    default:
        error->all(FLERR,
                "Unknown damping type\n"
                "\t\t'd3_damp_zero',\n"
                "\t\t'd3_damp_bj',\n"
                "\t\t'd3_damp_zerom',\n"
                "\t\t'd3_damp_bjm'\n"
                );
        break;
    }
}


/* ----------------------------------------------------------------------
   finds atomic number (used in PairD3::coeff)
------------------------------------------------------------------------- */

int PairD3::find_atomic_number(std::string& key) {
    std::transform(key.begin(), key.end(), key.begin(), ::tolower);
    if (key.length() == 1) { key += " "; }
    key.resize(2);

    std::vector<std::string> element_table = {
        "h ","he",
        "li","be","b ","c ","n ","o ","f ","ne",
        "na","mg","al","si","p ","s ","cl","ar",
        "k ","ca","sc","ti","v ","cr","mn","fe","co","ni","cu",
        "zn","ga","ge","as","se","br","kr",
        "rb","sr","y ","zr","nb","mo","tc","ru","rh","pd","ag",
        "cd","in","sn","sb","te","i ","xe",
        "cs","ba","la","ce","pr","nd","pm","sm","eu","gd","tb","dy",
        "ho","er","tm","yb","lu","hf","ta","w ","re","os","ir","pt",
        "au","hg","tl","pb","bi","po","at","rn",
        "fr","ra","ac","th","pa","u ","np","pu"
    };

    for (size_t i = 0; i < element_table.size(); ++i) {
        if (element_table[i] == key) {
            int atomic_number = i + 1;
            return atomic_number;
        }
    }

    // if not the case
    return -1;
}

/* ----------------------------------------------------------------------
   Check whether an integer value in an integer array (used in PairD3::coeff)
------------------------------------------------------------------------- */

int PairD3::is_int_in_array(int arr[], int size, int value) {
    for (int i = 0; i < size; i++) {
        if (arr[i] == value) { return i; } // returns the index
    }
    return -1;
}

/* ----------------------------------------------------------------------
   Read r0ab values from r0ab.csv (used in PairD3::coeff)
------------------------------------------------------------------------- */

void PairD3::read_r0ab(LAMMPS* lmp, char* path_r0ab, int* atomic_numbers, int ntypes) {

    int nparams_per_line = 94;
    int row_idx = 1;
    char* line;

    PotentialFileReader r0ab_reader(lmp, path_r0ab, "d3");

    while ((line = r0ab_reader.next_line(nparams_per_line))) {
        const int idx_atom_1 = is_int_in_array(atomic_numbers, ntypes, row_idx);
        // Skip for the other rows
        if (idx_atom_1 < 0) { row_idx++; continue; }
        try {
            ValueTokenizer r0ab_values(line);

            for (int col_idx=1; col_idx <= nparams_per_line; col_idx++) {
                const double value = r0ab_values.next_double();
                const int idx_atom_2 = is_int_in_array(atomic_numbers, ntypes, col_idx);
                if (idx_atom_2 < 0) { continue; }
                r0ab[idx_atom_1+1][idx_atom_2+1] = value / AU_TO_ANG;
            } // loop over column

            row_idx++;
        } catch (TokenizerException& e) {
            error->one(FLERR, e.what());
        } // loop over rows
    }
}

/* ----------------------------------------------------------------------
   Get atom pair indices and grid indices (used in PairD3::read_c6ab)
------------------------------------------------------------------------- */

void PairD3::get_limit_in_pars_array(int& idx_atom_1, int& idx_atom_2, int& idx_i, int& idx_j) {
    idx_i = 1;
    idx_j = 1;
    int shift = 100;

    while (idx_atom_1 > shift) {
        idx_atom_1 -= shift;
        idx_i++;
    }

    while (idx_atom_2 > shift) {
        idx_atom_2 -= shift;
        idx_j++;
    }
}

/* ----------------------------------------------------------------------
   Read c6ab values from c6ab.csv (used in PairD3::coeff)
------------------------------------------------------------------------- */

void PairD3::read_c6ab(LAMMPS* lmp, char* path_c6ab, int* atomic_numbers, int ntypes) {

    for (int i = 1; i <= ntypes; i++) { mxc[i] = 0; }

    int grid_i = 0, grid_j = 0;
    char* line;
    int nparams_per_line = 5;

    PotentialFileReader c6ab_reader(lmp, path_c6ab, "d3");

    while ((line = c6ab_reader.next_line(nparams_per_line))) {
        try {
            ValueTokenizer c6ab_values(line);
            const double ref_c6 = c6ab_values.next_double();
            int atom_number_1 = static_cast<int>(c6ab_values.next_double());
            int atom_number_2 = static_cast<int>(c6ab_values.next_double());
            get_limit_in_pars_array(atom_number_1, atom_number_2, grid_i, grid_j);
            const int idx_atom_1 = is_int_in_array(atomic_numbers, ntypes, atom_number_1);
            if ( idx_atom_1 < 0 ) { continue; }
            const int idx_atom_2 = is_int_in_array(atomic_numbers, ntypes, atom_number_2);
            if ( idx_atom_2 < 0 ) { continue; }
            const double ref_cn1 = c6ab_values.next_double();
            const double ref_cn2 = c6ab_values.next_double();

            mxc[idx_atom_1 + 1] = std::max(mxc[idx_atom_1 + 1], grid_i);
            mxc[idx_atom_2 + 1] = std::max(mxc[idx_atom_2 + 1], grid_j);
            c6ab[idx_atom_1 + 1][idx_atom_2 + 1][grid_i - 1][grid_j - 1][0] = ref_c6;
            c6ab[idx_atom_1 + 1][idx_atom_2 + 1][grid_i - 1][grid_j - 1][1] = ref_cn1;
            c6ab[idx_atom_1 + 1][idx_atom_2 + 1][grid_i - 1][grid_j - 1][2] = ref_cn2;
            c6ab[idx_atom_2 + 1][idx_atom_1 + 1][grid_j - 1][grid_i - 1][0] = ref_c6;
            c6ab[idx_atom_2 + 1][idx_atom_1 + 1][grid_j - 1][grid_i - 1][1] = ref_cn2;
            c6ab[idx_atom_2 + 1][idx_atom_1 + 1][grid_j - 1][grid_i - 1][2] = ref_cn1;

        } catch (TokenizerException& e) {
            error->one(FLERR, e.what());
        } // loop over rows
    }

}

/* ----------------------------------------------------------------------
   Set functional parameters (used in PairD3::coeff)
------------------------------------------------------------------------- */

void PairD3::setfuncpar(char* functional_name) {
    // set parameters for the given functionals
    int zero_damping = 1;
    int bj_damping = 2;
    int zero_damping_modified = 3;
    int bj_damping_modified = 4;

    if (damping_type == zero_damping) {
        s6 = 1.0;
        alp = 14.0;
        rs18 = 1.0;

        // default def2-QZVP (almost basis set limit)
        std::unordered_map<std::string, int> commandMap = {
        { "slater-dirac-exchange", 1}, { "b-lyp", 2 },    { "b-p", 3 },       { "b97-d", 4 },      { "revpbe", 5 },
        { "pbe", 6 },                  { "pbesol", 7 },   { "rpw86-pbe", 8 }, { "rpbe", 9 },       { "tpss", 10 },
        { "b3-lyp", 11 },              { "pbe0", 12 },    { "hse06", 13 },    { "revpbe38", 14 },  { "pw6b95", 15 },
        { "tpss0", 16 },               { "b2-plyp", 17 }, { "pwpb95", 18 },   { "b2gp-plyp", 19 }, { "ptpss", 20 },
        { "hf", 21 },                  { "mpwlyp", 22 },  { "bpbe", 23 },     { "bh-lyp", 24 },    { "tpssh", 25 },
        { "pwb6k", 26 },               { "b1b95", 27 },   { "bop", 28 },      { "o-lyp", 29 },     { "o-pbe", 30 },
        { "ssb", 31 },                 { "revssb", 32 },  { "otpss", 33 },    { "b3pw91", 34 },    { "revpbe0", 35 },
        { "pbe38", 36 },               { "mpw1b95", 37 }, { "mpwb1k", 38 },   { "bmk", 39 },       { "cam-b3lyp", 40 },
        { "lc-wpbe", 41 },             { "m05", 42 },     { "m052x", 43 },    { "m06l", 44 },      { "m06", 45 },
        { "m062x", 46 },               { "m06hf", 47 },   { "hcth120", 48 }
        };

        int commandCode = commandMap[functional_name];
        switch (commandCode) {
        case 1: rs6 = 0.999; s18 = -1.957; rs18 = 0.697; break;
        case 2: rs6 = 1.094; s18 = 1.682; break;
        case 3: rs6 = 1.139; s18 = 1.683; break;
        case 4: rs6 = 0.892; s18 = 0.909; break;
        case 5: rs6 = 0.923; s18 = 1.010; break;
        case 6: rs6 = 1.217; s18 = 0.722; break;
        case 7: rs6 = 1.345; s18 = 0.612; break;
        case 8: rs6 = 1.224; s18 = 0.901; break;
        case 9: rs6 = 0.872; s18 = 0.514; break;
        case 10: rs6 = 1.166; s18 = 1.105; break;
        case 11: rs6 = 1.261; s18 = 1.703; break;
        case 12: rs6 = 1.287; s18 = 0.928; break;
        case 13: rs6 = 1.129; s18 = 0.109; break;
        case 14: rs6 = 1.021; s18 = 0.862; break;
        case 15: rs6 = 1.532; s18 = 0.862; break;
        case 16: rs6 = 1.252; s18 = 1.242; break;
        case 17: rs6 = 1.427; s18 = 1.022; s6 = 0.64; break;
        case 18: rs6 = 1.557; s18 = 0.705; s6 = 0.82; break;
        case 19: rs6 = 1.586; s18 = 0.760; s6 = 0.56; break;
        case 20: rs6 = 1.541; s18 = 0.879; s6 = 0.75; break;
        case 21: rs6 = 1.158; s18 = 1.746; break;
        case 22: rs6 = 1.239; s18 = 1.098; break;
        case 23: rs6 = 1.087; s18 = 2.033; break;
        case 24: rs6 = 1.370; s18 = 1.442; break;
        case 25: rs6 = 1.223; s18 = 1.219; break;
        case 26: rs6 = 1.660; s18 = 0.550; break;
        case 27: rs6 = 1.613; s18 = 1.868; break;
        case 28: rs6 = 0.929; s18 = 1.975; break;
        case 29: rs6 = 0.806; s18 = 1.764; break;
        case 30: rs6 = 0.837; s18 = 2.055; break;
        case 31: rs6 = 1.215; s18 = 0.663; break;
        case 32: rs6 = 1.221; s18 = 0.560; break;
        case 33: rs6 = 1.128; s18 = 1.494; break;
        case 34: rs6 = 1.176; s18 = 1.775; break;
        case 35: rs6 = 0.949; s18 = 0.792; break;
        case 36: rs6 = 1.333; s18 = 0.998; break;
        case 37: rs6 = 1.605; s18 = 1.118; break;
        case 38: rs6 = 1.671; s18 = 1.061; break;
        case 39: rs6 = 1.931; s18 = 2.168; break;
        case 40: rs6 = 1.378; s18 = 1.217; break;
        case 41: rs6 = 1.355; s18 = 1.279; break;
        case 42: rs6 = 1.373; s18 = 0.595; break;
        case 43: rs6 = 1.417; s18 = 0.000; break;
        case 44: rs6 = 1.581; s18 = 0.000; break;
        case 45: rs6 = 1.325; s18 = 0.000; break;
        case 46: rs6 = 1.619; s18 = 0.000; break;
        case 47: rs6 = 1.446; s18 = 0.000; break;
        /* DFTB3(zeta = 4.0), old deprecated parameters; case ("dftb3"); rs6 = 1.235; s18 = 0.673; */
        case 48: rs6 = 1.221; s18 = 1.206; break;
        default:
            error->all(FLERR, "Functional name unknown");
            break;
        }

    } else if (damping_type == bj_damping) {
        s6 = 1.0;
        alp = 14.0;

        std::unordered_map<std::string, int> commandMap = {
            {"b-p", 1}, {"b-lyp", 2}, {"revpbe", 3}, {"rpbe", 4}, {"b97-d", 5}, {"pbe", 6},
            {"rpw86-pbe", 7}, {"b3-lyp", 8}, {"tpss", 9}, {"hf", 10}, {"tpss0", 11}, {"pbe0", 12},
            {"hse06", 13}, {"revpbe38", 14}, {"pw6b95", 15}, {"b2-plyp", 16}, {"dsd-blyp", 17},
            {"dsd-blyp-fc", 18}, {"bop", 19}, {"mpwlyp", 20}, {"o-lyp", 21}, {"pbesol", 22}, {"bpbe", 23},
            {"opbe", 24}, {"ssb", 25}, {"revssb", 26}, {"otpss", 27}, {"b3pw91", 28}, {"bh-lyp", 29},
            {"revpbe0", 30}, {"tpssh", 31}, {"mpw1b95", 32}, {"pwb6k", 33}, {"b1b95", 34}, {"bmk", 35},
            {"cam-b3lyp", 36}, {"lc-wpbe", 37}, {"b2gp-plyp", 38}, {"ptpss", 39}, {"pwpb95", 40},
            {"hf/mixed", 41}, {"hf/sv", 42}, {"hf/minis", 43}, {"b3-lyp/6-31gd", 44}, {"hcth120", 45},
            {"pw1pw", 46}, {"pwgga", 47}, {"hsesol", 48}, {"hf3c", 49}, {"hf3cv", 50}, {"pbeh3c", 51},
            {"pbeh-3c", 52}
        };

        int commandCode = commandMap[functional_name];
        switch (commandCode) {
            case 1: rs6 = 0.3946; s18 = 3.2822; rs18 = 4.8516; break;
            case 2: rs6 = 0.4298; s18 = 2.6996; rs18 = 4.2359; break;
            case 3: rs6 = 0.5238; s18 = 2.3550; rs18 = 3.5016; break;
            case 4: rs6 = 0.1820; s18 = 0.8318; rs18 = 4.0094; break;
            case 5: rs6 = 0.5545; s18 = 2.2609; rs18 = 3.2297; break;
            case 6: rs6 = 0.4289; s18 = 0.7875; rs18 = 4.4407; break;
            case 7: rs6 = 0.4613; s18 = 1.3845; rs18 = 4.5062; break;
            case 8: rs6 = 0.3981; s18 = 1.9889; rs18 = 4.4211; break;
            case 9: rs6 = 0.4535; s18 = 1.9435; rs18 = 4.4752; break;
            case 10: rs6 = 0.3385; s18 = 0.9171; rs18 = 2.8830; break;
            case 11: rs6 = 0.3768; s18 = 1.2576; rs18 = 4.5865; break;
            case 12: rs6 = 0.4145; s18 = 1.2177; rs18 = 4.8593; break;
            case 13: rs6 = 0.383; s18 = 2.310; rs18 = 5.685; break;
            case 14: rs6 = 0.4309; s18 = 1.4760; rs18 = 3.9446; break;
            case 15: rs6 = 0.2076; s18 = 0.7257; rs18 = 6.3750; break;
            case 16: rs6 = 0.3065; s18 = 0.9147; rs18 = 5.0570; break; s6 = 0.64;
            case 17: rs6 = 0.0000; s18 = 0.2130; rs18 = 6.0519; s6 = 0.50; break;
            case 18: rs6 = 0.0009; s18 = 0.2112; rs18 = 5.9807; s6 = 0.50; break;
            case 19: rs6 = 0.4870; s18 = 3.2950; rs18 = 3.5043; break;
            case 20: rs6 = 0.4831; s18 = 2.0077; rs18 = 4.5323; break;
            case 21: rs6 = 0.5299; s18 = 2.6205; rs18 = 2.8065; break;
            case 22: rs6 = 0.4466; s18 = 2.9491; rs18 = 6.1742; break;
            case 23: rs6 = 0.4567; s18 = 4.0728; rs18 = 4.3908; break;
            case 24: rs6 = 0.5512; s18 = 3.3816; rs18 = 2.9444; break;
            case 25: rs6 = -0.0952; s18 = -0.1744; rs18 = 5.2170; break;
            case 26: rs6 = 0.4720; s18 = 0.4389; rs18 = 4.0986; break;
            case 27: rs6 = 0.4634; s18 = 2.7495; rs18 = 4.3153; break;
            case 28: rs6 = 0.4312; s18 = 2.8524; rs18 = 4.4693; break;
            case 29: rs6 = 0.2793; s18 = 1.0354; rs18 = 4.9615; break;
            case 30: rs6 = 0.4679; s18 = 1.7588; rs18 = 3.7619; break;
            case 31: rs6 = 0.4529; s18 = 2.2382; rs18 = 4.6550; break;
            case 32: rs6 = 0.1955; s18 = 1.0508; rs18 = 6.4177; break;
            case 33: rs6 = 0.1805; s18 = 0.9383; rs18 = 7.7627; break;
            case 34: rs6 = 0.2092; s18 = 1.4507; rs18 = 5.5545; break;
            case 35: rs6 = 0.1940; s18 = 2.0860; rs18 = 5.9197; break;
            case 36: rs6 = 0.3708; s18 = 2.0674; rs18 = 5.4743; break;
            case 37: rs6 = 0.3919; s18 = 1.8541; rs18 = 5.0897; break;
            case 38: rs6 = 0.0000; s18 = 0.2597; rs18 = 6.3332; s6 = 0.560; break;
            case 39: rs6 = 0.0000; s18 = 0.2804; rs18 = 6.5745; s6 = 0.750; break;
            case 40: rs6 = 0.0000; s18 = 0.2904; rs18 = 7.3141; s6 = 0.820; break;
            // special HF / DFT with eBSSE correction;
            case 41: rs6 = 0.5607; s18 = 3.9027; rs18 = 4.5622; break;
            case 42: rs6 = 0.4249; s18 = 2.1849; rs18 = 4.2783; break;
            case 43: rs6 = 0.1702; s18 = 0.9841; rs18 = 3.8506; break;
            case 44: rs6 = 0.5014; s18 = 4.0672; rs18 = 4.8409; break;
            case 45: rs6 = 0.3563; s18 = 1.0821; rs18 = 4.3359; break;
            /*     DFTB3 old, deprecated parameters : ;
             *     case ("dftb3"); rs6 = 0.7461; s18 = 3.209; rs18 = 4.1906;
             *     special SCC - DFTB parametrization;
             *     full third order DFTB, self consistent charges, hydrogen pair damping with; exponent 4.2;
            */
            case 46: rs6 = 0.3807; s18 = 2.3363; rs18 = 5.8844; break;
            case 47: rs6 = 0.2211; s18 = 2.6910; rs18 = 6.7278; break;
            case 48: rs6 = 0.4650; s18 = 2.9215; rs18 = 6.2003; break;
            // special HF - D3 - gCP - SRB / MINIX parametrization;
            case 49: rs6 = 0.4171; s18 = 0.8777; rs18 = 2.9149; break;
            // special HF - D3 - gCP - SRB2 / ECP - 2G parametrization;
            case 50: rs6 = 0.3063; s18 = 0.5022; rs18 = 3.9856; break;
            // special PBEh - D3 - gCP / def2 - mSVP parametrization;
            case 51: rs6 = 0.4860; s18 = 0.0000; rs18 = 4.5000; break;
            case 52: rs6 = 0.4860; s18 = 0.0000; rs18 = 4.5000; break;
            default:
                error->all(FLERR, "Functional name unknown");
                break;
        }
    } else if (damping_type == zero_damping_modified) {
        s6 = 1.0;
        alp = 14.0;

        std::unordered_map<std::string, int> commandMap = {
            {"b2-plyp", 1}, {"b3-lyp", 2}, {"b97-d", 3}, {"b-lyp", 4},
            {"b-p", 5}, {"pbe", 6}, {"pbe0", 7}, {"lc-wpbe", 8}
        };

        int commandCode = commandMap[functional_name];
        switch (commandCode) {
            case 1: rs6 = 1.313134; s18 = 0.717543; rs18 = 0.016035; s6 = 0.640000; break;
            case 2: rs6 = 1.338153; s18 = 1.532981; rs18 = 0.013988; break;
            case 3: rs6 = 1.151808; s18 = 1.020078; rs18 = 0.035964; break;
            case 4: rs6 = 1.279637; s18 = 1.841686; rs18 = 0.014370; break;
            case 5: rs6 = 1.233460; s18 = 1.945174; rs18 = 0.000000; break;
            case 6: rs6 = 2.340218; s18 = 0.000000; rs18 = 0.129434; break;
            case 7: rs6 = 2.077949; s18 = 0.000081; rs18 = 0.116755; break;
            case 8: rs6 = 1.366361; s18 = 1.280619; rs18 = 0.003160; break;
            default:
                error->all(FLERR, "Functional name unknown");
                break;
        }
    } else if (damping_type == bj_damping_modified) {
        // BJ damping
        s6 = 1.0;
        alp = 14.0;

        std::unordered_map<std::string, int> commandMap = {
            {"b2-plyp", 1}, {"b3-lyp", 2}, {"b97-d", 3}, {"b-lyp", 4},
            {"b-p", 5}, {"pbe", 6}, {"pbe0", 7}, {"lc-wpbe", 8}
        };

        int commandCode = commandMap[functional_name];
        switch (commandCode) {
            case 1: rs6 = 0.486434; s18 = 0.672820; rs18 = 3.656466; s6 = 0.640000; break;
            case 2: rs6 = 0.278672; s18 = 1.466677; rs18 = 4.606311; break;
            case 3: rs6 = 0.240184; s18 = 1.206988; rs18 = 3.864426; break;
            case 4: rs6 = 0.448486; s18 = 1.875007; rs18 = 3.610679; break;
            case 5: rs6 = 0.821850; s18 = 3.140281; rs18 = 2.728151; break;
            case 6: rs6 = 0.012092; s18 = 0.358940; rs18 = 5.938951; break;
            case 7: rs6 = 0.007912; s18 = 0.528823; rs18 = 6.162326; break;
            case 8: rs6 = 0.563761; s18 = 0.906564; rs18 = 3.593680; break;
            default:
                error->all(FLERR, "Functional name unknown");
                break;
        }
    } else {
        error->all(FLERR, "Unknown damping type");
    }

    rs8 = rs18;
    alp6 = alp;
    alp8 = alp + 2.0;

}

/* ----------------------------------------------------------------------
   Coeff: read from pair_coeff (Required)
          pair_coeff * * path_r0ab.csv path_c6ab.csv functional element1 element2 ...
------------------------------------------------------------------------- */

void PairD3::coeff(int narg, char **arg) {
    if (!allocated) allocate();
    if (narg < 3) { error->all(FLERR, "Pair_coeff * * needs: r0ab.csv c6ab.csv functional element1 element2 ..."); }

    std::string element;
    int ntypes = atom->ntypes;
    int* atomic_numbers = (int*)malloc(sizeof(int)*ntypes);
    for (int i = 0; i < ntypes; i++) {
        element = arg[i+5];
        atomic_numbers[i] = find_atomic_number(element);
    }

    int count = 0;
    for (int i = 1; i <= atom->ntypes; i++) {
        for (int j = 1; j <= atom->ntypes; j++) {
            setflag[i][j] = 1;
            count++;
        }
    }

    if (count == 0) error->all(FLERR,"Incorrect args for pair coefficients");

    /*
    scale r4/r2 values of the atoms by sqrt(Z)
    sqrt is also globally close to optimum
    together with the factor 1/2 this yield reasonable
    c8 for he, ne and ar. for larger Z, C8 becomes too large
    which effectively mimics higher R^n terms neglected due
    to stability reasons

    r2r4 =sqrt(0.5*r2r4(i)*dfloat(i)**0.5 ) with i=elementnumber
    the large number of digits is just to keep the results consistent
    with older versions. They should not imply any higher accuracy than
    the old values
    */
    double r2r4_ref[94] = {
         2.00734898,  1.56637132,  5.01986934,  3.85379032,  3.64446594,
         3.10492822,  2.71175247,  2.59361680,  2.38825250,  2.21522516,
         6.58585536,  5.46295967,  5.65216669,  4.88284902,  4.29727576,
         4.04108902,  3.72932356,  3.44677275,  7.97762753,  7.07623947,
         6.60844053,  6.28791364,  6.07728703,  5.54643096,  5.80491167,
         5.58415602,  5.41374528,  5.28497229,  5.22592821,  5.09817141,
         6.12149689,  5.54083734,  5.06696878,  4.87005108,  4.59089647,
         4.31176304,  9.55461698,  8.67396077,  7.97210197,  7.43439917,
         6.58711862,  6.19536215,  6.01517290,  5.81623410,  5.65710424,
         5.52640661,  5.44263305,  5.58285373,  7.02081898,  6.46815523,
         5.98089120,  5.81686657,  5.53321815,  5.25477007, 11.02204549,
        10.15679528,  9.35167836,  9.06926079,  8.97241155,  8.90092807,
         8.85984840,  8.81736827,  8.79317710,  7.89969626,  8.80588454,
         8.42439218,  8.54289262,  8.47583370,  8.45090888,  8.47339339,
         7.83525634,  8.20702843,  7.70559063,  7.32755997,  7.03887381,
         6.68978720,  6.05450052,  5.88752022,  5.70661499,  5.78450695,
         7.79780729,  7.26443867,  6.78151984,  6.67883169,  6.39024318,
         6.09527958, 11.79156076, 11.10997644,  9.51377795,  8.67197068,
         8.77140725,  8.65402716,  8.53923501,  8.85024712
    }; // atomic <r^2>/<r^4> values

    /*
    covalent radii (taken from Pyykko and Atsumi, Chem. Eur. J. 15, 2009, 188-197)
    values for metals decreased by 10 %
    !      data rcov/
    !     .  0.32, 0.46, 1.20, 0.94, 0.77, 0.75, 0.71, 0.63, 0.64, 0.67
    !     ., 1.40, 1.25, 1.13, 1.04, 1.10, 1.02, 0.99, 0.96, 1.76, 1.54
    !     ., 1.33, 1.22, 1.21, 1.10, 1.07, 1.04, 1.00, 0.99, 1.01, 1.09
    !     ., 1.12, 1.09, 1.15, 1.10, 1.14, 1.17, 1.89, 1.67, 1.47, 1.39
    !     ., 1.32, 1.24, 1.15, 1.13, 1.13, 1.08, 1.15, 1.23, 1.28, 1.26
    !     ., 1.26, 1.23, 1.32, 1.31, 2.09, 1.76, 1.62, 1.47, 1.58, 1.57
    !     ., 1.56, 1.55, 1.51, 1.52, 1.51, 1.50, 1.49, 1.49, 1.48, 1.53
    !     ., 1.46, 1.37, 1.31, 1.23, 1.18, 1.16, 1.11, 1.12, 1.13, 1.32
    !     ., 1.30, 1.30, 1.36, 1.31, 1.38, 1.42, 2.01, 1.81, 1.67, 1.58
    !     ., 1.52, 1.53, 1.54, 1.55 /

    these new data are scaled with k2=4./3.  and converted a_0 via
    autoang=0.52917726d0
    */

    double rcov_ref[94] = {
        0.80628308, 1.15903197, 3.02356173, 2.36845659, 1.94011865,
        1.88972601, 1.78894056, 1.58736983, 1.61256616, 1.68815527,
        3.52748848, 3.14954334, 2.84718717, 2.62041997, 2.77159820,
        2.57002732, 2.49443835, 2.41884923, 4.43455700, 3.88023730,
        3.35111422, 3.07395437, 3.04875805, 2.77159820, 2.69600923,
        2.62041997, 2.51963467, 2.49443835, 2.54483100, 2.74640188,
        2.82199085, 2.74640188, 2.89757982, 2.77159820, 2.87238349,
        2.94797246, 4.76210950, 4.20778980, 3.70386304, 3.50229216,
        3.32591790, 3.12434702, 2.89757982, 2.84718717, 2.84718717,
        2.72120556, 2.89757982, 3.09915070, 3.22513231, 3.17473967,
        3.17473967, 3.09915070, 3.32591790, 3.30072128, 5.26603625,
        4.43455700, 4.08180818, 3.70386304, 3.98102289, 3.95582657,
        3.93062995, 3.90543362, 3.80464833, 3.82984466, 3.80464833,
        3.77945201, 3.75425569, 3.75425569, 3.72905937, 3.85504098,
        3.67866672, 3.45189952, 3.30072128, 3.09915070, 2.97316878,
        2.92277614, 2.79679452, 2.82199085, 2.84718717, 3.32591790,
        3.27552496, 3.27552496, 3.42670319, 3.30072128, 3.47709584,
        3.57788113, 5.06446567, 4.56053862, 4.20778980, 3.98102289,
        3.82984466, 3.85504098, 3.88023730, 3.90543362
    }; // covalent radii

    for (int i = 0; i < ntypes; i++) {
        r2r4[i+1] = r2r4_ref[atomic_numbers[i]-1];
        rcov[i+1] = rcov_ref[atomic_numbers[i]-1];
    }

    // set r0ab
    read_r0ab(lmp, arg[2], atomic_numbers, ntypes);

    // read c6ab
    read_c6ab(lmp, arg[3], atomic_numbers, ntypes);

    // read functional parameters
    setfuncpar(arg[4]);

    free(atomic_numbers);

}

/* ----------------------------------------------------------------------
   Get derivative of C6 w.r.t. CN (used in PairD3::compute)

   C6 = C6(CN_A, CN_B) == W(CN_A, CN_B) / Z(CN_A, CN_B)

   This gives below from chain rule:
   d(C6)/dr = d(C6)/d(CN_A) * d(CN_A)/dr + d(C6)/d(CN_B) * d(CN_B)/dr

   So we can pre-calculate the d(C6)/d(CN_A), d(C6)/d(CN_B) part.

   d(C6)/d(CN_i) = (dW/d(CN_i) * Z - W * dZ/d(CN_i)) / (W * W)
        W : "denominator"
        Z : "numerator"
        dW/d(CN_i) : "d_denominator_i"
        dZ/d(CN_j) : "d_numerator_j"

    Z = Sum( L_ij(CN_A, CN_B) * C6_ref(CN_A_i, CN_B_j) ) over i, j
    W = Sum( L_ij(CN_A, CN_B) ) over i, j

   And the resulting derivative term is saved into
   "dc6_iji_tot", "dc6_ijj_tot" array,
   where we can find the value of d(C6)/d(CN_i)
   by knowing the index of "iat", and "jat". ("idx_linij")

   Also, c6 values will also be saved into "c6_ij_tot" array.

   Here, as we only interested in *pair* of atoms, assume "iat" >= "jat".
   Then "idx_linij" = "jat + (iat + 1) * iat / 2" have the order below.

     idx_linij | j = 0  j = 1  j = 2  j = 3    ...
---------------------------------------------
        i = 0  |     0
        i = 1  |     1      2
        i = 2  |     3      4      5
        i = 3  |     6      7      8      9
          ...  |    ...    ...    ...    ...   ...

------------------------------------------------------------------------- */

__global__ void kernel_getdC6dCNij(
    int linij,
    int *type, double *cn, int *mxc, double *****c6ab, 
    double *c6_ij_tot, double *dc6_iji_tot, double *dc6_ijj_tot
) {

    int iter = blockIdx.x * blockDim.x + threadIdx.x;
    if (iter >= linij) return;

    int iat, jat;
    ij_at_linij(iter, iat, jat);

    const double cni = cn[iat];
    const int mxci = mxc[type[iat]];

    const double cnj = cn[jat];
    const int mxcj = mxc[type[jat]];

    double c6mem = -1e99;
    double r_save = 9999.0;
    double numerator = 0.0;
    double denominator = 0.0;
    double d_numerator_i = 0.0;
    double d_denominator_i = 0.0;
    double d_numerator_j = 0.0;
    double d_denominator_j = 0.0;

    const int idx_linij = iter;

    for (int a = 0; a < mxci; a++) {
        for (int b = 0; b < mxcj; b++) {
            const double c6ref = c6ab[type[iat]][type[jat]][a][b][0];

            if (c6ref > 0) {
                const double cn_refi = c6ab[type[iat]][type[jat]][a][b][1];
                const double cn_refj = c6ab[type[iat]][type[jat]][a][b][2];

                const double r = (cn_refi - cni) * (cn_refi - cni) + (cn_refj - cnj) * (cn_refj - cnj);
                if (r < r_save) {
                    r_save = r;
                    c6mem = c6ref;
                }

                double expterm = exp(-4.0 * r);
                numerator += c6ref * expterm;
                denominator += expterm;

                expterm *= 2.0 * -4.0;

                double term = expterm * (cni - cn_refi);
                d_numerator_i += c6ref * term;
                d_denominator_i += term;

                term = expterm * (cnj - cn_refj);
                d_numerator_j += c6ref * term;
                d_denominator_j += term;
            }
        }
    }

    if (denominator > 1e-99) {
        c6_ij_tot[idx_linij] = numerator / denominator;
        dc6_iji_tot[idx_linij] = ((d_numerator_i * denominator) - (d_denominator_i * numerator)) / (denominator * denominator);
        dc6_ijj_tot[idx_linij] = ((d_numerator_j * denominator) - (d_denominator_j * numerator)) / (denominator * denominator);
    }
    else {
        c6_ij_tot[idx_linij] = c6mem;
        dc6_iji_tot[idx_linij] = 0.0;
        dc6_ijj_tot[idx_linij] = 0.0;
    }
}

void PairD3::get_dC6_dCNij() {
    int n = atom->natoms;
    int linij = n * (n + 1) / 2;

    int *cuda_type;
    hipMallocManaged(&cuda_type, n * sizeof(int));
    hipMemcpy(cuda_type, atom->type, n * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 128;
    int blocksPerGrid = (linij + threadsPerBlock - 1) / threadsPerBlock;
    kernel_getdC6dCNij<<<blocksPerGrid, threadsPerBlock>>>(
        linij, cuda_type, cn, mxc, c6ab, c6_ij_tot, dc6_iji_tot, dc6_ijj_tot
    );
    hipDeviceSynchronize();

    hipFree(cuda_type);
}

/* ----------------------------------------------------------------------
   Get lattice vectors (used in PairD3::compute)

   1) Save lattice vectors into "lat_v_1", "lat_v_2", "lat_v_3"
   2) Calculate repetition criteria for vdw, cn
   3) precaluclate tau (xyz shift due to cell repetition)

------------------------------------------------------------------------- */

void PairD3::set_lattice_vectors() {

    double boxxlo = domain->boxlo[0];
    double boxxhi = domain->boxhi[0];
    double boxylo = domain->boxlo[1];
    double boxyhi = domain->boxhi[1];
    double boxzlo = domain->boxlo[2];
    double boxzhi = domain->boxhi[2];
    double xy = domain->xy;
    double xz = domain->xz;
    double yz = domain->yz;

    lat_v_1[0] = (boxxhi - boxxlo) / AU_TO_ANG;
    lat_v_1[1] =               0.0;
    lat_v_1[2] =               0.0;
    lat_v_2[0] =                xy / AU_TO_ANG;
    lat_v_2[1] = (boxyhi - boxylo) / AU_TO_ANG;
    lat_v_2[2] =               0.0;
    lat_v_3[0] =                xz / AU_TO_ANG;
    lat_v_3[1] =                yz / AU_TO_ANG;
    lat_v_3[2] = (boxzhi - boxzlo) / AU_TO_ANG;

    set_lattice_repetition_criteria(rthr, rep_vdw);
    set_lattice_repetition_criteria(cn_thr, rep_cn);

    int vdw_range_x = 2 * rep_vdw[0] + 1;
    int vdw_range_y = 2 * rep_vdw[1] + 1;
    int vdw_range_z = 2 * rep_vdw[2] + 1;
    int tau_loop_size_vdw = vdw_range_x * vdw_range_y * vdw_range_z * 3;
    if (tau_loop_size_vdw != tau_idx_vdw_total_size) {
        if (tau_idx_vdw != nullptr) {
            for (int i = 0; i < vdw_range_x; i++) {
                for (int j = 0; j < vdw_range_y; j++) {
                    for (int k = 0; k < vdw_range_z; k++) {
                        hipFree(tau_vdw[i][j][k]);
                    }
                    hipFree(tau_vdw[i][j]);
                }
                hipFree(tau_vdw[i]);
            }
            hipFree(tau_vdw);
            hipFree(tau_idx_vdw);
        }
        tau_idx_vdw_total_size = tau_loop_size_vdw;
        hipMallocManaged(&tau_vdw, vdw_range_x * sizeof(double***));
        for (int i = 0; i < vdw_range_x; i++) {
            hipMallocManaged(&tau_vdw[i], vdw_range_y * sizeof(double**));
            for (int j = 0; j < vdw_range_y; j++) {
                hipMallocManaged(&tau_vdw[i][j], vdw_range_z * sizeof(double*));
                for (int k = 0; k < vdw_range_z; k++) {
                    hipMallocManaged(&tau_vdw[i][j][k], 3 * sizeof(double));
                }
            }
        }
        hipMallocManaged(&tau_idx_vdw, tau_idx_vdw_total_size * sizeof(double));
    }

    int cn_range_x  = 2 * rep_cn[0] + 1;
    int cn_range_y  = 2 * rep_cn[1] + 1;
    int cn_range_z  = 2 * rep_cn[2] + 1;
    int tau_loop_size_cn = cn_range_x * cn_range_y * cn_range_z * 3;
    if (tau_loop_size_cn != tau_idx_cn_total_size) {
        if (tau_idx_cn != nullptr) {
            for (int i = 0; i < cn_range_x; i++) {
                for (int j = 0; j < cn_range_y; j++) {
                    for (int k = 0; k < cn_range_z; k++) {
                        hipFree(tau_cn[i][j][k]);
                    }
                    hipFree(tau_cn[i][j]);
                }
                hipFree(tau_cn[i]);
            }
            hipFree(tau_cn);
            hipFree(tau_idx_cn);
        }
        tau_idx_cn_total_size = tau_loop_size_cn;
        hipMallocManaged(&tau_cn, cn_range_x * sizeof(double***));
        for (int i = 0; i < cn_range_x; i++) {
            hipMallocManaged(&tau_cn[i], cn_range_y * sizeof(double**));
            for (int j = 0; j < cn_range_y; j++) {
                hipMallocManaged(&tau_cn[i][j], cn_range_z * sizeof(double*));
                for (int k = 0; k < cn_range_z; k++) {
                    hipMallocManaged(&tau_cn[i][j][k], 3 * sizeof(double));
                }
            }
        }
        hipMallocManaged(&tau_idx_cn, tau_idx_cn_total_size * sizeof(double));
    }

}

/* ----------------------------------------------------------------------
   Set repetition criteria (used in PairD3::compute)

   Needed as Periodic Boundary Condition should be considered.

   As the cell may *not* be orthorhombic,
   the dot product should be used between x/y/z direction and
   corresponding cross product vector.
------------------------------------------------------------------------- */

void PairD3::set_lattice_repetition_criteria(double r_threshold, int* rep_v) {
    double r_cutoff = sqrt(r_threshold);
    double lat_cp_12[3], lat_cp_23[3], lat_cp_31[3];
    double cos_value;

    MathExtra::cross3(lat_v_1, lat_v_2, lat_cp_12);
    MathExtra::cross3(lat_v_2, lat_v_3, lat_cp_23);
    MathExtra::cross3(lat_v_3, lat_v_1, lat_cp_31);

    cos_value = MathExtra::dot3(lat_cp_23, lat_v_1) / MathExtra::len3(lat_cp_23);
    rep_v[0] = static_cast<int>(std::abs(r_cutoff / cos_value)) + 1;
    cos_value = MathExtra::dot3(lat_cp_31, lat_v_2) / MathExtra::len3(lat_cp_31);
    rep_v[1] = static_cast<int>(std::abs(r_cutoff / cos_value)) + 1;
    cos_value = MathExtra::dot3(lat_cp_12, lat_v_3) / MathExtra::len3(lat_cp_12);
    rep_v[2] = static_cast<int>(std::abs(r_cutoff / cos_value)) + 1;

    if (domain->xperiodic == 0) { rep_v[0] = 0; }
    if (domain->yperiodic == 0) { rep_v[1] = 0; }
    if (domain->zperiodic == 0) { rep_v[2] = 0; }
}

/* ----------------------------------------------------------------------
   Calculate Coordination Number (used in PairD3::compute)
------------------------------------------------------------------------- */

__global__ void kernel_getCoordNumber(
    int linij, int maxtau,
    double **x, int *tau_idx_cn, double ****tau_cn, int *type, double *rcov, int *rep_cn, double cn_thr,
    double *cn
) {

    int iter = blockIdx.x * blockDim.x + threadIdx.x;
    if (iter >= linij) return;

    int iat, jat;
    ij_at_linij(iter, iat, jat);

    for (int k = maxtau - 1; k >= 0; k -= 3) {

        const int idx1 = tau_idx_cn[k-2];
        const int idx2 = tau_idx_cn[k-1];
        const int idx3 = tau_idx_cn[k];

        if (iat == jat) {
            if (idx1 != rep_cn[0] || idx2 != rep_cn[1] || idx3 != rep_cn[2]) {
                const double rx = tau_cn[idx1][idx2][idx3][0];
                const double ry = tau_cn[idx1][idx2][idx3][1];
                const double rz = tau_cn[idx1][idx2][idx3][2];
                const double r2 = rx * rx + ry * ry + rz * rz;
                if (r2 <= cn_thr) {
                    const double r = sqrt(r2);
                    const double damp = 1.0 / (1.0 + exp(-16.0 * (((rcov[type[iat]] + rcov[type[iat]]) / r) - 1.0)));
                    atomicAdd(&cn[iat], damp);
                }
            }
        } 
        
        else {
            const double rx = x[jat][0] - x[iat][0] + tau_cn[idx1][idx2][idx3][0];
            const double ry = x[jat][1] - x[iat][1] + tau_cn[idx1][idx2][idx3][1];
            const double rz = x[jat][2] - x[iat][2] + tau_cn[idx1][idx2][idx3][2];
            const double r2 = rx * rx + ry * ry + rz * rz;
            if (r2 <= cn_thr) {
                const double r = sqrt(r2);
                const double damp = 1.0 / (1.0 + exp(-16.0 * (((rcov[type[iat]] + rcov[type[jat]]) / r) - 1.0)));
                atomicAdd(&cn[iat], damp);
                atomicAdd(&cn[jat], damp);
            }
        }

    }
}

void PairD3::get_coordination_number() {
    int n = atom->natoms;
    int linij = n * (n + 1) / 2;
    int maxtau = tau_idx_cn_total_size;

    // for (int i = 0; i < n; i++) {
    //     cn[i] = 0.0;
    // }
    hipMemset(cn, 0, n * sizeof(double));

    int *cuda_type;
    hipMallocManaged(&cuda_type, n * sizeof(int));
    hipMemcpy(cuda_type, atom->type, n * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 128;
    int blocksPerGrid = (linij + threadsPerBlock - 1) / threadsPerBlock;

    kernel_getCoordNumber<<<blocksPerGrid, threadsPerBlock>>>(
        linij, maxtau, x, tau_idx_cn, tau_cn, cuda_type, rcov, rep_cn, cn_thr, cn
    );
    hipDeviceSynchronize();

    hipFree(cuda_type);

    get_dC6_dCNij();
}

/* ----------------------------------------------------------------------
   reallcate memory if the number of atoms has changed (used in PairD3::compute)
------------------------------------------------------------------------- */

void PairD3::reallocate_arrays() {

    int n = atom->natoms;

    /* -------------- Destroy previous arrays -------------- */

    hipFree(cn); for (int i = 0; i < n; i++) { hipFree(x[i]); }
    hipFree(x);
    hipFree(dc6i); for (int i = 0; i < n; i++) { hipFree(f[i]); }
    hipFree(f);
    hipFree(dc6_iji_tot);
    hipFree(dc6_ijj_tot);
    hipFree(c6_ij_tot);

    /* -------------- Destroy previous arrays -------------- */

    /* -------------- Create new arrays -------------- */
    n_save = n;

    hipMallocManaged(&cn, n * sizeof(double));
    hipMallocManaged(&x, n * sizeof(double*));
    for (int i = 0; i < n; i++) { hipMallocManaged(&x[i], 3 * sizeof(double)); }
    hipMallocManaged(&dc6i, n * sizeof(double));
    hipMallocManaged(&f, n * sizeof(double*));
    for (int i = 0; i < n; i++) { hipMallocManaged(&f[i], 3 * sizeof(double)); }

    set_lattice_vectors();

    int n_ij_combination = n * (n + 1) / 2;
    hipMallocManaged(&dc6_iji_tot, n_ij_combination * sizeof(double));
    hipMallocManaged(&dc6_ijj_tot, n_ij_combination * sizeof(double));
    hipMallocManaged(&c6_ij_tot,   n_ij_combination * sizeof(double));

    //int vdw_range_x = 2 * rep_vdw[0] + 1;
    //int vdw_range_y = 2 * rep_vdw[1] + 1;
    //int vdw_range_z = 2 * rep_vdw[2] + 1;
    //tau_idx_vdw_total_size = vdw_range_x * vdw_range_y * vdw_range_z * 3;

    //int cn_range_x  = 2 * rep_cn[0] + 1;
    //int cn_range_y  = 2 * rep_cn[1] + 1;
    //int cn_range_z  = 2 * rep_cn[2] + 1;
    //tau_idx_cn_total_size = cn_range_x * cn_range_y * cn_range_z * 3;

    //hipMallocManaged(&tau_vdw, vdw_range_x * sizeof(double***));
    //for (int i = 0; i < vdw_range_x; i++) {
    //    hipMallocManaged(&tau_vdw[i], vdw_range_y * sizeof(double**));
    //    for (int j = 0; j < vdw_range_y; j++) {
    //        hipMallocManaged(&tau_vdw[i][j], vdw_range_z * sizeof(double*));
    //        for (int k = 0; k < vdw_range_z; k++) {
    //            hipMallocManaged(&tau_vdw[i][j][k], 3 * sizeof(double));
    //        }
    //    }
    //}
    //hipMallocManaged(&tau_idx_vdw, tau_idx_vdw_total_size * sizeof(int));

    //hipMallocManaged(&tau_cn, cn_range_x * sizeof(double***));
    //for (int i = 0; i < cn_range_x; i++) {
    //    hipMallocManaged(&tau_cn[i], cn_range_y * sizeof(double**));
    //    for (int j = 0; j < cn_range_y; j++) {
    //        hipMallocManaged(&tau_cn[i][j], cn_range_z * sizeof(double*));
    //        for (int k = 0; k < cn_range_z; k++) {
    //            hipMallocManaged(&tau_cn[i][j][k], 3 * sizeof(double));
    //        }
    //    }
    //}
    //hipMallocManaged(&tau_idx_cn, tau_idx_cn_total_size * sizeof(int));

    /* -------------- Create new arrays -------------- */
}

/* ----------------------------------------------------------------------
  Initialize atomic positions & types (used in PairD3::compute)

   As the default xyz from lammps does not assure that atoms are within unit cell,
   this function shifts atoms into the unit cell.
------------------------------------------------------------------------- */

void PairD3::load_atom_info() {
    double lat[3][3];
    lat[0][0] = lat_v_1[0];
    lat[0][1] = lat_v_2[0];
    lat[0][2] = lat_v_3[0];
    lat[1][0] = lat_v_1[1];
    lat[1][1] = lat_v_2[1];
    lat[1][2] = lat_v_3[1];
    lat[2][0] = lat_v_1[2];
    lat[2][1] = lat_v_2[2];
    lat[2][2] = lat_v_3[2];

    double det = lat[0][0] * lat[1][1] * lat[2][2]
               + lat[0][1] * lat[1][2] * lat[2][0]
               + lat[0][2] * lat[1][0] * lat[2][1]
               - lat[0][2] * lat[1][1] * lat[2][0]
               - lat[0][1] * lat[1][0] * lat[2][2]
               - lat[0][0] * lat[1][2] * lat[2][1];

    double lat_inv[3][3];
    lat_inv[0][0] = (lat[1][1] * lat[2][2] - lat[1][2] * lat[2][1]) / det;
    lat_inv[1][0] = (lat[1][2] * lat[2][0] - lat[1][0] * lat[2][2]) / det;
    lat_inv[2][0] = (lat[1][0] * lat[2][1] - lat[1][1] * lat[2][0]) / det;
    lat_inv[0][1] = (lat[0][2] * lat[2][1] - lat[0][1] * lat[2][2]) / det;
    lat_inv[1][1] = (lat[0][0] * lat[2][2] - lat[0][2] * lat[2][0]) / det;
    lat_inv[2][1] = (lat[0][1] * lat[2][0] - lat[0][0] * lat[2][1]) / det;
    lat_inv[0][2] = (lat[0][1] * lat[1][2] - lat[0][2] * lat[1][1]) / det;
    lat_inv[1][2] = (lat[0][2] * lat[1][0] - lat[0][0] * lat[1][2]) / det;
    lat_inv[2][2] = (lat[0][0] * lat[1][1] - lat[0][1] * lat[1][0]) / det;

    double a[3] = { 0.0 };
    for (int iat = 0; iat < atom->natoms; iat++) {
        for (int i = 0; i < 3; i++) {
            a[i] = lat_inv[i][0] * (atom->x)[iat][0] + lat_inv[i][1] * (atom->x)[iat][1] + lat_inv[i][2] * (atom->x)[iat][2];
            if      (a[i] > 1) { while (a[i] > 1) { a[i]--; } }
            else if (a[i] < 0) { while (a[i] < 0) { a[i]++; } }
        }

        for (int i = 0; i < 3; i++) {
            x[iat][i] = (lat[i][0] * a[0] + lat[i][1] * a[1] + lat[i][2] * a[2]) / AU_TO_ANG;
        }
    }
}

/* ----------------------------------------------------------------------
   Precalculate tau array
------------------------------------------------------------------------- */

void PairD3::precalculate_tau_array() {
    int xlim = rep_vdw[0];
    int ylim = rep_vdw[1];
    int zlim = rep_vdw[2];

    int index = 0;
    for (int taux = -xlim; taux <= xlim; taux++) {
        for (int tauy = -ylim; tauy <= ylim; tauy++) {
            for (int tauz = -zlim; tauz <= zlim; tauz++) {
                tau_vdw[taux + xlim][tauy + ylim][tauz + zlim][0] = lat_v_1[0] * taux + lat_v_2[0] * tauy + lat_v_3[0] * tauz;
                tau_vdw[taux + xlim][tauy + ylim][tauz + zlim][1] = lat_v_1[1] * taux + lat_v_2[1] * tauy + lat_v_3[1] * tauz;
                tau_vdw[taux + xlim][tauy + ylim][tauz + zlim][2] = lat_v_1[2] * taux + lat_v_2[2] * tauy + lat_v_3[2] * tauz;
                tau_idx_vdw[index++] = taux + xlim;
                tau_idx_vdw[index++] = tauy + ylim;
                tau_idx_vdw[index++] = tauz + zlim;
            }
        }
    }

    xlim = rep_cn[0];
    ylim = rep_cn[1];
    zlim = rep_cn[2];

    index = 0;
    for (int taux = -xlim; taux <= xlim; taux++) {
        for (int tauy = -ylim; tauy <= ylim; tauy++) {
            for (int tauz = -zlim; tauz <= zlim; tauz++) {
                tau_cn[taux + xlim][tauy + ylim][tauz + zlim][0] = lat_v_1[0] * taux + lat_v_2[0] * tauy + lat_v_3[0] * tauz;
                tau_cn[taux + xlim][tauy + ylim][tauz + zlim][1] = lat_v_1[1] * taux + lat_v_2[1] * tauy + lat_v_3[1] * tauz;
                tau_cn[taux + xlim][tauy + ylim][tauz + zlim][2] = lat_v_1[2] * taux + lat_v_2[2] * tauy + lat_v_3[2] * tauz;
                tau_idx_cn[index++] = taux + xlim;
                tau_idx_cn[index++] = tauy + ylim;
                tau_idx_cn[index++] = tauz + zlim;
            }
        }
    }
}


/* ----------------------------------------------------------------------
   Get forces (Zero damping)
------------------------------------------------------------------------- */

__global__ void kernel_getForcesWithoutZero(
    int linij, int maxtau,
    double s6, double s8, double a1_sqrt3, double a1, double a2, double r2_rthr, double alp6, double alp8,
    double **x, int *type, double *dc6i, double *r2r4, double **r0ab, int *tau_idx_vdw, double ****tau_vdw, int *rep_vdw,
    double *c6_ij_tot, double *dc6_iji_tot, double *dc6_ijj_tot, 
    double *disp, double **f, double **sigma
) {

    int iter = blockIdx.x * blockDim.x + threadIdx.x;

    // for block reduction

    __shared__ double sigma_00[128];
    __shared__ double sigma_01[128];
    __shared__ double sigma_02[128];
    __shared__ double sigma_10[128];
    __shared__ double sigma_11[128];
    __shared__ double sigma_12[128];
    __shared__ double sigma_20[128];
    __shared__ double sigma_21[128];
    __shared__ double sigma_22[128];
    __shared__ double disp_shared[128];

    // for private threads
    double sigma_local_00 = 0.0;
    double sigma_local_01 = 0.0;
    double sigma_local_02 = 0.0;
    double sigma_local_10 = 0.0;
    double sigma_local_11 = 0.0;
    double sigma_local_12 = 0.0;
    double sigma_local_20 = 0.0;
    double sigma_local_21 = 0.0;
    double sigma_local_22 = 0.0;
    double disp_local = 0.0;

    if (iter < linij) {
        
        int iat, jat;
        ij_at_linij(iter, iat, jat);

        for (int k = maxtau - 1; k >= 0; k -= 3) {

            const int idx1 = tau_idx_vdw[k-2];
            const int idx2 = tau_idx_vdw[k-1];
            const int idx3 = tau_idx_vdw[k];

            if (iat == jat) {

                if (idx1 == rep_vdw[0] && idx2 == rep_vdw[1] && idx3 == rep_vdw[2]) { continue; }
                const double rij[3] = {
                    tau_vdw[idx1][idx2][idx3][0],
                    tau_vdw[idx1][idx2][idx3][1],
                    tau_vdw[idx1][idx2][idx3][2]
                };
                const double r2 = lensq3(rij);

                if (r2 > r2_rthr || r2 < 0.1) { continue; }

                const double r2_inv = 1.0 / r2;
                const double r = sqrt(r2);
                const double r_inv = 1.0 / r;
                const double r0 = r0ab[type[iat]][type[iat]];

                double tmp_v = (a1 * r0) * r_inv;
                tmp_v *= tmp_v * tmp_v * tmp_v * tmp_v * tmp_v * tmp_v; // ^7
                double t6 = tmp_v * tmp_v; // ^14
                const double damp6 = 1.0 / (1.0 + 6.0 * t6);
                tmp_v = (a2 * r0) * r_inv;
                tmp_v = tmp_v * tmp_v; // ^2
                tmp_v = tmp_v * tmp_v; // ^4
                tmp_v = tmp_v * tmp_v; // ^8
                double t8 = tmp_v * tmp_v; // ^16
                const double damp8 = 1.0 / (1.0 + 6.0 * t8);

                const double c6 = c6_ij_tot[iter];
                const double r42 = r2r4[type[iat]] * r2r4[type[iat]];
                const double r6_inv = r2_inv * r2_inv * r2_inv;
                const double r7_inv = r6_inv * r_inv;
                const double x1 = 0.5 * 6.0 * c6 * r7_inv * (s6 * damp6 * (alp6 * t6 * damp6 - 1.0) + s8 * r42 * r2_inv * damp8 * (3.0 * alp8 * t8 * damp8 - 4.0)) * r_inv;

                const double vec[3] = {
                    x1 * rij[0],
                    x1 * rij[1],
                    x1 * rij[2]
                };

                sigma_local_00 += vec[0] * rij[0];
                sigma_local_01 += vec[0] * rij[1];
                sigma_local_02 += vec[0] * rij[2];
                sigma_local_10 += vec[1] * rij[0];
                sigma_local_11 += vec[1] * rij[1];
                sigma_local_12 += vec[1] * rij[2];
                sigma_local_20 += vec[2] * rij[0];
                sigma_local_21 += vec[2] * rij[1];
                sigma_local_22 += vec[2] * rij[2];

                const double dc6_rest = (s6 * damp6 + 3.0 * s8 * r42 * damp8 * r2_inv) * r6_inv * 0.5;
                disp_local -= dc6_rest * c6;
                const double dc6iji = dc6_iji_tot[iter];
                const double dc6ijj = dc6_ijj_tot[iter];
                atomicAdd(&dc6i[iat], dc6_rest * dc6iji * 2);
            }
            
            else {
                const double rij[3] = {
                    x[jat][0] - x[iat][0] + tau_vdw[idx1][idx2][idx3][0],
                    x[jat][1] - x[iat][1] + tau_vdw[idx1][idx2][idx3][1],
                    x[jat][2] - x[iat][2] + tau_vdw[idx1][idx2][idx3][2]
                };
                const double r2 = lensq3(rij);

                if (r2 > r2_rthr || r2 < 0.1) { continue; }

                const double r2_inv = 1.0 / r2;
                const double r = sqrt(r2);
                const double r_inv = 1.0 / r;
                const double r0 = r0ab[type[iat]][type[jat]];

                double tmp_v = (a1 * r0) * r_inv;
                double t6 = tmp_v;
                t6 *= t6;       // ^2
                t6 *= tmp_v;    // ^3
                t6 *= t6;       // ^6
                t6 *= tmp_v;    // ^7
                t6 *= t6;       // ^14
                const double damp6 = 1.0 / (1.0 + 6.0 * t6);
                t6 *= damp6;    // pre-calculation
                double t8 = (a2 * r0) * r_inv;
                t8 *= t8;       // ^2
                t8 *= t8;       // ^4
                t8 *= t8;       // ^8
                t8 *= t8;       // ^16
                const double damp8 = 1.0 / (1.0 + 6.0 * t8);
                t8 *= damp8;    // pre-calculation

                const double c6 = c6_ij_tot[iter];
                const double r6_inv = r2_inv * r2_inv * r2_inv;
                const double r7_inv = r6_inv * r_inv;

                const double r42 = r2r4[type[iat]] * r2r4[type[jat]];
                /* // d(r ^ (-6)) / d(r_ij) */
                const double x1 = 6.0 * c6 * r7_inv * (s6 * damp6 * (14.0 * t6 - 1.0) + s8 * r42 * r2_inv * damp8 * (48.0 * t8 - 4.0)) * r_inv;

                const double vec[3] = {
                    x1 * rij[0],
                    x1 * rij[1],
                    x1 * rij[2]
                };

                atomicAdd(&f[iat][0], -vec[0]);
                atomicAdd(&f[iat][1], -vec[1]);
                atomicAdd(&f[iat][2], -vec[2]);
                atomicAdd(&f[jat][0], vec[0]);
                atomicAdd(&f[jat][1], vec[1]);
                atomicAdd(&f[jat][2], vec[2]);

                sigma_local_00 += vec[0] * rij[0];
                sigma_local_01 += vec[0] * rij[1];
                sigma_local_02 += vec[0] * rij[2];
                sigma_local_10 += vec[1] * rij[0];
                sigma_local_11 += vec[1] * rij[1];
                sigma_local_12 += vec[1] * rij[2];
                sigma_local_20 += vec[2] * rij[0];
                sigma_local_21 += vec[2] * rij[1];
                sigma_local_22 += vec[2] * rij[2];

                const double dc6_rest = (s6 * damp6 + 3.0 * s8 * r42 * damp8 * r2_inv) * r6_inv;
                disp_local -= dc6_rest * c6;
                const double dc6iji = dc6_iji_tot[iter];
                const double dc6ijj = dc6_ijj_tot[iter];
                atomicAdd(&dc6i[iat], dc6_rest * dc6iji);
                atomicAdd(&dc6i[jat], dc6_rest * dc6ijj);
            }
        }

    }

    // save to shared memory
    sigma_00[threadIdx.x] = sigma_local_00;
    sigma_01[threadIdx.x] = sigma_local_01;
    sigma_02[threadIdx.x] = sigma_local_02;
    sigma_10[threadIdx.x] = sigma_local_10;
    sigma_11[threadIdx.x] = sigma_local_11;
    sigma_12[threadIdx.x] = sigma_local_12;
    sigma_20[threadIdx.x] = sigma_local_20;
    sigma_21[threadIdx.x] = sigma_local_21;
    sigma_22[threadIdx.x] = sigma_local_22;
    disp_shared[threadIdx.x] = disp_local;
    __syncthreads();

    // reduction
    for (int s=blockDim.x/2; s>0; s>>=1) {
        if (threadIdx.x < s) {
            sigma_00[threadIdx.x] += sigma_00[threadIdx.x + s];
            sigma_01[threadIdx.x] += sigma_01[threadIdx.x + s];
            sigma_02[threadIdx.x] += sigma_02[threadIdx.x + s];
            sigma_10[threadIdx.x] += sigma_10[threadIdx.x + s];
            sigma_11[threadIdx.x] += sigma_11[threadIdx.x + s];
            sigma_12[threadIdx.x] += sigma_12[threadIdx.x + s];
            sigma_20[threadIdx.x] += sigma_20[threadIdx.x + s];
            sigma_21[threadIdx.x] += sigma_21[threadIdx.x + s];
            sigma_22[threadIdx.x] += sigma_22[threadIdx.x + s];
            disp_shared[threadIdx.x] += disp_shared[threadIdx.x + s];
        }
        __syncthreads();
    }

    // save to global memory
    if (threadIdx.x == 0) {
        atomicAdd(&sigma[0][0], sigma_00[0]);
        atomicAdd(&sigma[0][1], sigma_01[0]);
        atomicAdd(&sigma[0][2], sigma_02[0]);
        atomicAdd(&sigma[1][0], sigma_10[0]);
        atomicAdd(&sigma[1][1], sigma_11[0]);
        atomicAdd(&sigma[1][2], sigma_12[0]);
        atomicAdd(&sigma[2][0], sigma_20[0]);
        atomicAdd(&sigma[2][1], sigma_21[0]);
        atomicAdd(&sigma[2][2], sigma_22[0]);
        atomicAdd(disp, disp_shared[0]);
    }

}

void PairD3::get_forces_without_dC6_zero_damping() {
    int n = atom->natoms;
    int np1 = atom->ntypes + 1;
    int linij = n * (n + 1) / 2;
    int maxtau = tau_idx_vdw_total_size;

    for (int dim = 0; dim < n; dim++) { dc6i[dim] = 0.0; }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < 3; j++) {
            f[i][j] = 0.0;
        }
    }

    for (int ii = 0; ii < 3; ii++) {
        for (int jj = 0; jj < 3; jj++) {
            sigma[ii][jj] = 0.0;
        }
    }

    const double s8 = s18;
    const double a1 = rs6;
    const double a1_sqrt3 = a1 * sqrt(3);
    const double a2 = rs8;
    const double r2_rthr = rthr;

    double *cuda_disp;
    hipMallocManaged(&cuda_disp, sizeof(double));
    *cuda_disp = 0.0;

    int *cuda_type;
    hipMallocManaged(&cuda_type, sizeof(int) * n);
    hipMemcpy(cuda_type, atom->type, n * sizeof(int), hipMemcpyHostToDevice);

    double *cuda_r2r4;
    hipMallocManaged(&cuda_r2r4, sizeof(double) * np1);
    hipMemcpy(cuda_r2r4, r2r4, np1 * sizeof(double), hipMemcpyHostToDevice);

    double **cuda_r0ab;
    hipMallocManaged(&cuda_r0ab, sizeof(double*) * np1);
    for (int i = 0; i < np1; i++) {
        hipMallocManaged(&cuda_r0ab[i], sizeof(double) * np1);
    }
    for (int i = 0; i < np1; i++) {
        hipMemcpy(cuda_r0ab[i], r0ab[i], np1 * sizeof(double), hipMemcpyHostToDevice);
    }

    int threadsPerBlock = 128;
    int blocksPerGrid = (linij + threadsPerBlock - 1) / threadsPerBlock;

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start);

    kernel_getForcesWithoutZero<<<blocksPerGrid, threadsPerBlock>>>(
        linij, maxtau, s6, s8, a1_sqrt3, a1, a2, r2_rthr, alp6, alp8, x, cuda_type, dc6i, cuda_r2r4, cuda_r0ab, tau_idx_vdw, tau_vdw, rep_vdw, c6_ij_tot, dc6_iji_tot, dc6_ijj_tot, cuda_disp, f, sigma
    );
    hipDeviceSynchronize();

    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Time elapsed for get_forces_without_dC6_bj_damping: %f ms\n", milliseconds);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);

    hipFree(cuda_type);
    hipFree(cuda_r2r4);
    for (int i = 0; i < np1; i++) {
        hipFree(cuda_r0ab[i]);
    }
    hipFree(cuda_r0ab);

    disp_total = *cuda_disp;
    hipFree(cuda_disp);

}

/* ----------------------------------------------------------------------
   Get forces (Zero damping)
------------------------------------------------------------------------- */

// openACC not implemented
void PairD3::get_forces_without_dC6_zero_damping_modified() {

}

/* ----------------------------------------------------------------------
   Get forces (BJ damping)
------------------------------------------------------------------------- */

__global__ void kernel_getForcesWithoutBJ(
    int linij, int maxtau,
    double s6, double s8, double a1_sqrt3, double a2, double r2_rthr, 
    double **x, int *type, double *dc6i, double *r2r4, int *tau_idx_vdw, double ****tau_vdw, int *rep_vdw,
    double *c6_ij_tot, double *dc6_iji_tot, double *dc6_ijj_tot, 
    double *disp, double **f, double **sigma
) {

    int iter = blockIdx.x * blockDim.x + threadIdx.x;

    // for block reduction
    __shared__ double sigma_00[128];
    __shared__ double sigma_01[128];
    __shared__ double sigma_02[128];
    __shared__ double sigma_10[128];
    __shared__ double sigma_11[128];
    __shared__ double sigma_12[128];
    __shared__ double sigma_20[128];
    __shared__ double sigma_21[128];
    __shared__ double sigma_22[128];
    __shared__ double disp_shared[128];

    // for private threads
    double sigma_local_00 = 0.0;
    double sigma_local_01 = 0.0;
    double sigma_local_02 = 0.0;
    double sigma_local_10 = 0.0;
    double sigma_local_11 = 0.0;
    double sigma_local_12 = 0.0;
    double sigma_local_20 = 0.0;
    double sigma_local_21 = 0.0;
    double sigma_local_22 = 0.0;
    double disp_local = 0.0;

    if (iter < linij) {

        int iat, jat;
        ij_at_linij(iter, iat, jat);

        for (int k = maxtau - 1; k >= 0; k -= 3) {

            const int idx1 = tau_idx_vdw[k-2];
            const int idx2 = tau_idx_vdw[k-1];
            const int idx3 = tau_idx_vdw[k];

            if (iat == jat) {

                if (idx1 == rep_vdw[0] && idx2 == rep_vdw[1] && idx3 == rep_vdw[2]) { continue; }
                const double rij[3] = {
                    tau_vdw[idx1][idx2][idx3][0],
                    tau_vdw[idx1][idx2][idx3][1],
                    tau_vdw[idx1][idx2][idx3][2]
                };
                const double r2 = lensq3(rij);

                if (r2 > r2_rthr || r2 < 0.1) { continue; }

                const double r = sqrt(r2);
                const double r4 = r2 * r2;
                const double r6 = r4 * r2;
                const double r7 = r6 * r;
                const double r8 = r4 * r4;

                const double r42 = r2r4[type[iat]] * r2r4[type[iat]];
                const double R0 = a1_sqrt3 * sqrt(r42) + a2;
                const double R0_6 = R0 * R0 * R0 * R0 * R0 * R0;
                const double R0_8 = R0 * R0 * R0 * R0 * R0 * R0 * R0 * R0;
                const double t6 = r6 + R0_6;
                const double t8 = r8 + R0_8;

                const double c6 = c6_ij_tot[iter];

                const double t6_squared_inv = 1.0 / (t6 * t6);
                const double t8_squared_inv = 1.0 / (t8 * t8);

                const double x1 =\
                    0.5 * (- s6 * c6 *  6.0 * r4 * r * t6_squared_inv
                            - s8 * c6 * 24.0 * r42 * r7 * t8_squared_inv);

                const double r_inv = 1.0 / r;
                const double vec[3] = {
                    x1 * rij[0] * r_inv,
                    x1 * rij[1] * r_inv,
                    x1 * rij[2] * r_inv
                };

                sigma_local_00 += vec[0] * rij[0];
                sigma_local_01 += vec[0] * rij[1];
                sigma_local_02 += vec[0] * rij[2];
                sigma_local_10 += vec[1] * rij[0];
                sigma_local_11 += vec[1] * rij[1];
                sigma_local_12 += vec[1] * rij[2];
                sigma_local_20 += vec[2] * rij[0];
                sigma_local_21 += vec[2] * rij[1];
                sigma_local_22 += vec[2] * rij[2];

                const double dc6_rest = (s6 / t6 + 3.0 * s8 * r42 / t8) * 0.5;
                disp_local -= dc6_rest * c6;
                const double dc6iji = dc6_iji_tot[iter];
                const double dc6ijj = dc6_ijj_tot[iter];
                atomicAdd(&dc6i[iat], dc6_rest * dc6iji * 2);
            }
            
            else {
                const double rij[3] = {
                    x[jat][0] - x[iat][0] + tau_vdw[idx1][idx2][idx3][0],
                    x[jat][1] - x[iat][1] + tau_vdw[idx1][idx2][idx3][1],
                    x[jat][2] - x[iat][2] + tau_vdw[idx1][idx2][idx3][2]
                };
                const double r2 = lensq3(rij);
                if (r2 > r2_rthr) { continue; }

                const double r = sqrt(r2);
                const double r4 = r2 * r2;
                const double r6 = r4 * r2;
                const double r7 = r6 * r;
                const double r8 = r4 * r4;

                // Calculates damping functions
                const double r42 = r2r4[type[iat]] * r2r4[type[jat]];
                const double R0 = a1_sqrt3 * sqrt(r42) + a2;
                const double R0_6 = R0 * R0 * R0 * R0 * R0 * R0;
                const double R0_8 = R0 * R0 * R0 * R0 * R0 * R0 * R0 * R0;
                const double t6 = r6 + R0_6;
                const double t8 = r8 + R0_8;

                const double c6 = c6_ij_tot[iter];

                const double t6_squared_inv = 1.0 / (t6 * t6);
                const double t8_squared_inv = 1.0 / (t8 * t8);

                /* // d(r ^ (-6)) / d(r_ij) */
                const double x1 = \
                    - s6 * c6 *  6.0 *  r4 * r  * t6_squared_inv
                    - s8 * c6 * 24.0 * r42 * r7 * t8_squared_inv;

                const double r_inv = 1.0 / r;
                const double vec[3] = {
                    x1 * rij[0] * r_inv,
                    x1 * rij[1] * r_inv,
                    x1 * rij[2] * r_inv
                };

                atomicAdd(&f[iat][0], -vec[0]);
                atomicAdd(&f[iat][1], -vec[1]);
                atomicAdd(&f[iat][2], -vec[2]);
                atomicAdd(&f[jat][0], vec[0]);
                atomicAdd(&f[jat][1], vec[1]);
                atomicAdd(&f[jat][2], vec[2]);

                sigma_local_00 += vec[0] * rij[0];
                sigma_local_01 += vec[0] * rij[1];
                sigma_local_02 += vec[0] * rij[2];
                sigma_local_10 += vec[1] * rij[0];
                sigma_local_11 += vec[1] * rij[1];
                sigma_local_12 += vec[1] * rij[2];
                sigma_local_20 += vec[2] * rij[0];
                sigma_local_21 += vec[2] * rij[1];
                sigma_local_22 += vec[2] * rij[2];

                // in dC6_rest all terms BUT C6 - term is saved for the kat - loop
                const double dc6_rest = s6 / t6 + 3.0 * s8 * r42 / t8;
                disp_local -= dc6_rest * c6;
                const double dc6iji = dc6_iji_tot[iter];
                const double dc6ijj = dc6_ijj_tot[iter];
                atomicAdd(&dc6i[iat], dc6_rest * dc6iji);
                atomicAdd(&dc6i[jat], dc6_rest * dc6ijj);
            }

        }

    }

    // save to shared memory
    sigma_00[threadIdx.x] = sigma_local_00;
    sigma_01[threadIdx.x] = sigma_local_01;
    sigma_02[threadIdx.x] = sigma_local_02;
    sigma_10[threadIdx.x] = sigma_local_10;
    sigma_11[threadIdx.x] = sigma_local_11;
    sigma_12[threadIdx.x] = sigma_local_12;
    sigma_20[threadIdx.x] = sigma_local_20;
    sigma_21[threadIdx.x] = sigma_local_21;
    sigma_22[threadIdx.x] = sigma_local_22;
    disp_shared[threadIdx.x] = disp_local;
    __syncthreads();

    // reduction
    for (int s=blockDim.x/2; s>0; s>>=1) {
        if (threadIdx.x < s) {
            sigma_00[threadIdx.x] += sigma_00[threadIdx.x + s];
            sigma_01[threadIdx.x] += sigma_01[threadIdx.x + s];
            sigma_02[threadIdx.x] += sigma_02[threadIdx.x + s];
            sigma_10[threadIdx.x] += sigma_10[threadIdx.x + s];
            sigma_11[threadIdx.x] += sigma_11[threadIdx.x + s];
            sigma_12[threadIdx.x] += sigma_12[threadIdx.x + s];
            sigma_20[threadIdx.x] += sigma_20[threadIdx.x + s];
            sigma_21[threadIdx.x] += sigma_21[threadIdx.x + s];
            sigma_22[threadIdx.x] += sigma_22[threadIdx.x + s];
            disp_shared[threadIdx.x] += disp_shared[threadIdx.x + s];
        }
        __syncthreads();
    }

    // save to global memory
    if (threadIdx.x == 0) {
        atomicAdd(&sigma[0][0], sigma_00[0]);
        atomicAdd(&sigma[0][1], sigma_01[0]);
        atomicAdd(&sigma[0][2], sigma_02[0]);
        atomicAdd(&sigma[1][0], sigma_10[0]);
        atomicAdd(&sigma[1][1], sigma_11[0]);
        atomicAdd(&sigma[1][2], sigma_12[0]);
        atomicAdd(&sigma[2][0], sigma_20[0]);
        atomicAdd(&sigma[2][1], sigma_21[0]);
        atomicAdd(&sigma[2][2], sigma_22[0]);
        atomicAdd(disp, disp_shared[0]);
    }

}

void PairD3::get_forces_without_dC6_bj_damping() {
    int n = atom->natoms;
    int np1 = atom->ntypes + 1;
    int linij = n * (n + 1) / 2;
    int maxtau = tau_idx_vdw_total_size;

    for (int dim = 0; dim < n; dim++) { dc6i[dim] = 0.0; }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < 3; j++) {
            f[i][j] = 0.0;
        }
    }

    for (int ii = 0; ii < 3; ii++) {
        for (int jj = 0; jj < 3; jj++) {
            sigma[ii][jj] = 0.0;
        }
    }

    const double s8 = s18;
    const double a1 = rs6;
    const double a1_sqrt3 = a1 * sqrt(3);
    const double a2 = rs8;
    const double r2_rthr = rthr;

    double *cuda_disp;
    hipMallocManaged(&cuda_disp, sizeof(double));
    *cuda_disp = 0.0;

    int *cuda_type;
    hipMallocManaged(&cuda_type, sizeof(int) * n);
    hipMemcpy(cuda_type, atom->type, n * sizeof(int), hipMemcpyHostToDevice);

    double *cuda_r2r4;
    hipMallocManaged(&cuda_r2r4, sizeof(double) * np1);
    hipMemcpy(cuda_r2r4, r2r4, np1 * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlock = 128;
    int blocksPerGrid = (linij + threadsPerBlock - 1) / threadsPerBlock;

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start);

    kernel_getForcesWithoutBJ<<<blocksPerGrid, threadsPerBlock>>>(
        linij, maxtau, s6, s8, a1_sqrt3, a2, r2_rthr, x, cuda_type, dc6i, cuda_r2r4, tau_idx_vdw, tau_vdw, rep_vdw, c6_ij_tot, dc6_iji_tot, dc6_ijj_tot, cuda_disp, f, sigma
    );
    hipDeviceSynchronize();

    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Time elapsed for get_forces_without_dC6_bj_damping: %f ms\n", milliseconds);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);

    hipFree(cuda_type);
    hipFree(cuda_r2r4);

    disp_total = *cuda_disp;
    hipFree(cuda_disp);

}

/* ----------------------------------------------------------------------
   Get forces
------------------------------------------------------------------------- */

__global__ void kernel_getForcesWith(
    int linij, int maxtau, 
    double *dc6i, double **x, int *type, double *rcov, double cn_thr, double ****tau_cn, int *tau_idx_cn, int *rep_cn,
    double **f, double **sigma
) {

    int iter = blockIdx.x * blockDim.x + threadIdx.x;

    // for block reduction
    __shared__ double sigma_00[128];
    __shared__ double sigma_01[128];
    __shared__ double sigma_02[128];
    __shared__ double sigma_10[128];
    __shared__ double sigma_11[128];
    __shared__ double sigma_12[128];
    __shared__ double sigma_20[128];
    __shared__ double sigma_21[128];
    __shared__ double sigma_22[128];

    // for private threads
    double sigma_local_00 = 0.0;
    double sigma_local_01 = 0.0;
    double sigma_local_02 = 0.0;
    double sigma_local_10 = 0.0;
    double sigma_local_11 = 0.0;
    double sigma_local_12 = 0.0;
    double sigma_local_20 = 0.0;
    double sigma_local_21 = 0.0;
    double sigma_local_22 = 0.0;

    if (iter < linij) {

        int iat, jat;
        ij_at_linij(iter, iat, jat);

        for (int k = maxtau - 1; k >= 0; k -= 3) {
                
            const int idx1 = tau_idx_cn[k-2];
            const int idx2 = tau_idx_cn[k-1];
            const int idx3 = tau_idx_cn[k];

            if (iat == jat) {
    
                if (idx1 == rep_cn[0] && idx2 == rep_cn[1] && idx3 == rep_cn[2]) { continue; }
                const double rij[3] = {
                    tau_cn[idx1][idx2][idx3][0],
                    tau_cn[idx1][idx2][idx3][1],
                    tau_cn[idx1][idx2][idx3][2],
                };
                const double r2 = lensq3(rij);
                // Assume rthr > cn_thr --> only check for cn_thr
                if (r2 >= cn_thr) { continue; }
                const double r = sqrt(r2);
                const double r_inv = 1.0 / r;
                const double rcovij = rcov[type[iat]] + rcov[type[iat]];
                const double expterm = exp(-16.0 * (rcovij * r_inv - 1.0));
                const double dcnn = -16.0 * rcovij * expterm / (r2 * (expterm + 1.0) * (expterm + 1.0));
                const double x1 = dcnn * dc6i[iat];

                const double vec[3] = {
                    x1 * rij[0] * r_inv,
                    x1 * rij[1] * r_inv,
                    x1 * rij[2] * r_inv
                };

                sigma_local_00 += vec[0] * rij[0];
                sigma_local_01 += vec[0] * rij[1];
                sigma_local_02 += vec[0] * rij[2];
                sigma_local_10 += vec[1] * rij[0];
                sigma_local_11 += vec[1] * rij[1];
                sigma_local_12 += vec[1] * rij[2];
                sigma_local_20 += vec[2] * rij[0];
                sigma_local_21 += vec[2] * rij[1];
                sigma_local_22 += vec[2] * rij[2];

            } 
            
            else {
                const double rij[3] = {
                    x[jat][0] - x[iat][0] + tau_cn[idx1][idx2][idx3][0],
                    x[jat][1] - x[iat][1] + tau_cn[idx1][idx2][idx3][1],
                    x[jat][2] - x[iat][2] + tau_cn[idx1][idx2][idx3][2]
                };
                const double r2 = lensq3(rij);
                // Assume rthr > cn_thr --> only check for cn_thr
                if (r2 >= cn_thr) { continue; }
                const double r = sqrt(r2);
                const double r_inv = 1.0 / r;
                const double rcovij = rcov[type[iat]] + rcov[type[jat]];
                const double expterm = exp(-16.0 * (rcovij * r_inv - 1.0));
                const double dcnn = -16.0 * rcovij * expterm / (r2 * (expterm + 1.0) * (expterm + 1.0));
                const double x1 = dcnn * (dc6i[iat] + dc6i[jat]);

                const double vec[3] = {
                    x1 * rij[0] * r_inv,
                    x1 * rij[1] * r_inv,
                    x1 * rij[2] * r_inv
                };

                atomicAdd(&f[iat][0], -vec[0]);
                atomicAdd(&f[iat][1], -vec[1]);
                atomicAdd(&f[iat][2], -vec[2]);
                atomicAdd(&f[jat][0], vec[0]);
                atomicAdd(&f[jat][1], vec[1]);
                atomicAdd(&f[jat][2], vec[2]);


                sigma_local_00 += vec[0] * rij[0];
                sigma_local_01 += vec[0] * rij[1];
                sigma_local_02 += vec[0] * rij[2];
                sigma_local_10 += vec[1] * rij[0];
                sigma_local_11 += vec[1] * rij[1];
                sigma_local_12 += vec[1] * rij[2];
                sigma_local_20 += vec[2] * rij[0];
                sigma_local_21 += vec[2] * rij[1];
                sigma_local_22 += vec[2] * rij[2];

            }

        }

    }

    // save to shared memory
    sigma_00[threadIdx.x] = sigma_local_00;
    sigma_01[threadIdx.x] = sigma_local_01;
    sigma_02[threadIdx.x] = sigma_local_02;
    sigma_10[threadIdx.x] = sigma_local_10;
    sigma_11[threadIdx.x] = sigma_local_11;
    sigma_12[threadIdx.x] = sigma_local_12;
    sigma_20[threadIdx.x] = sigma_local_20;
    sigma_21[threadIdx.x] = sigma_local_21;
    sigma_22[threadIdx.x] = sigma_local_22;
    __syncthreads();

    // reduction
    for (int s=blockDim.x/2; s>0; s>>=1) {
        if (threadIdx.x < s) {
            sigma_00[threadIdx.x] += sigma_00[threadIdx.x + s];
            sigma_01[threadIdx.x] += sigma_01[threadIdx.x + s];
            sigma_02[threadIdx.x] += sigma_02[threadIdx.x + s];
            sigma_10[threadIdx.x] += sigma_10[threadIdx.x + s];
            sigma_11[threadIdx.x] += sigma_11[threadIdx.x + s];
            sigma_12[threadIdx.x] += sigma_12[threadIdx.x + s];
            sigma_20[threadIdx.x] += sigma_20[threadIdx.x + s];
            sigma_21[threadIdx.x] += sigma_21[threadIdx.x + s];
            sigma_22[threadIdx.x] += sigma_22[threadIdx.x + s];
        }
        __syncthreads();
    }

    // save to global memory
    if (threadIdx.x == 0) {
        atomicAdd(&sigma[0][0], sigma_00[0]);
        atomicAdd(&sigma[0][1], sigma_01[0]);
        atomicAdd(&sigma[0][2], sigma_02[0]);
        atomicAdd(&sigma[1][0], sigma_10[0]);
        atomicAdd(&sigma[1][1], sigma_11[0]);
        atomicAdd(&sigma[1][2], sigma_12[0]);
        atomicAdd(&sigma[2][0], sigma_20[0]);
        atomicAdd(&sigma[2][1], sigma_21[0]);
        atomicAdd(&sigma[2][2], sigma_22[0]);
    }

}

void PairD3::get_forces_with_dC6() {
    int n = atom->natoms;
    int linij = n * (n + 1) / 2;
    int maxtau = tau_idx_cn_total_size;

    int *cuda_type;
    hipMallocManaged(&cuda_type, sizeof(int) * n);
    hipMemcpy(cuda_type, atom->type, n * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 128;
    int blocksPerGrid = (linij + threadsPerBlock - 1) / threadsPerBlock;
    kernel_getForcesWith<<<blocksPerGrid, threadsPerBlock>>>(
        linij, maxtau, dc6i, x, cuda_type, rcov, cn_thr, tau_cn, tau_idx_cn, rep_cn, f, sigma
    );
    hipDeviceSynchronize();

    hipFree(cuda_type);
}


/* ----------------------------------------------------------------------
   Update energy, force, and stress
------------------------------------------------------------------------- */

void PairD3::update(int eflag, int vflag) {
    int n = atom->natoms;
    // Energy update
    if (eflag) { eng_vdwl += disp_total * AU_TO_EV; }

    double** f_local = atom->f;       // Local force of atoms
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < 3; j++) {
            f_local[i][j] += f[i][j] * AU_TO_EV / AU_TO_ANG;
        }
    }

    // Stress update
    if (vflag) {
        virial[0] += sigma[0][0] * AU_TO_EV;
        virial[1] += sigma[1][1] * AU_TO_EV;
        virial[2] += sigma[2][2] * AU_TO_EV;
        virial[3] += sigma[0][1] * AU_TO_EV;
        virial[4] += sigma[0][2] * AU_TO_EV;
        virial[5] += sigma[1][2] * AU_TO_EV;
    }
}

/* ----------------------------------------------------------------------
   Compute : energy, force, and stress (Required)
------------------------------------------------------------------------- */

void PairD3::compute(int eflag, int vflag) {
    if (eflag || vflag)          { ev_setup(eflag, vflag); }
    if (atom->natoms != n_save)  { reallocate_arrays(); }

    set_lattice_vectors();
    precalculate_tau_array();
    load_atom_info();

    get_coordination_number();

    int zero_damping = 1;
    int zero_damping_modified = 3;

    if (damping_type == zero_damping) {
        get_forces_without_dC6_zero_damping();
    }
    else if (damping_type == zero_damping_modified){
        get_forces_without_dC6_zero_damping_modified();
    }
    else {
        get_forces_without_dC6_bj_damping();
    }
    get_forces_with_dC6();
    update(eflag, vflag);
    
}

/* ----------------------------------------------------------------------
   init for one type pair i,j and corresponding j,i
------------------------------------------------------------------------- */

double PairD3::init_one(int i, int j) {
    if (setflag[i][j] == 0) error->all(FLERR, "All pair coeffs are not set");
    // No need to count local neighbor in D3
    /* return std::sqrt(rthr * std::pow(au_to_ang, 2)); */
    return 0.0;
}

/* ----------------------------------------------------------------------
   init specific to this pair style (Optional)
------------------------------------------------------------------------- */

void PairD3::init_style() {
    neighbor->add_request(this, NeighConst::REQ_FULL);
}

/* ----------------------------------------------------------------------
   proc 0 writes to restart file
------------------------------------------------------------------------- */

void PairD3::write_restart(FILE *fp) {}

/* ----------------------------------------------------------------------
   proc 0 reads from restart file, bcasts
------------------------------------------------------------------------- */

void PairD3::read_restart(FILE *fp) {}

/* ----------------------------------------------------------------------
   proc 0 writes to restart file
------------------------------------------------------------------------- */

void PairD3::write_restart_settings(FILE *fp) {}

/* ----------------------------------------------------------------------
   proc 0 reads from restart file, bcasts
------------------------------------------------------------------------- */

void PairD3::read_restart_settings(FILE *fp) {}
